#include "hip/hip_runtime.h"
#include <torch/library.h>
// Copied and adapted from
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/native/cuda/UpSample.cuh>

// Below is experimental temporary code before merging it to PyTorch
namespace at {
namespace native {
namespace internal_upsample {

__device__ __forceinline__ size_t
idx(const size_t nc,
    const size_t height,
    const size_t width,
    const size_t y,
    const size_t x) {
  return (nc * height + y) * width + x;
}

// taken from
// https://github.com/python-pillow/Pillow/blob/6812205f18ca4ef54372e87e1a13ce4a859434df/
// src/libImaging/Resample.c#L20-L29
template <typename accscalar_t>
__device__ __forceinline__ static accscalar_t bilinear_filter(accscalar_t x) {
  if (x < 0.0) {
    x = -x;
  }
  if (x < 1.0) {
    return static_cast<accscalar_t>(1.0) - x;
  }
  return static_cast<accscalar_t>(0.0);
}

// taken from
// https://github.com/python-pillow/Pillow/blob/6812205f18ca4ef54372e87e1a13ce4a859434df/
// src/libImaging/Resample.c#L46-L62
template <typename accscalar_t>
__device__ __forceinline__ static accscalar_t bicubic_filter(accscalar_t x) {
  // https://en.wikipedia.org/wiki/Bicubic_interpolation#Bicubic_convolution_algorithm
#define a -0.5
  if (x < 0.0) {
    x = -x;
  }
  if (x < 1.0) {
    return ((a + 2.0) * x - (a + 3.0)) * x * x + static_cast<accscalar_t>(1.0);
  }
  if (x < 2.0) {
    return (((x - 5) * x + 8) * x - 4) * a;
  }
  return static_cast<accscalar_t>(0.0);
#undef a
}

template <typename scalar_t, typename accscalar_t, typename filter_fn_t>
__device__ __forceinline__ static void _compute_weights(
    const int64_t i,
    const int64_t input_size,
    const accscalar_t scale,
    const accscalar_t support,
    scalar_t* wt_ptr,
    int64_t interp_size,
    filter_fn_t filter_fn,
    int64_t& xmin,
    int64_t& xmax) {
  accscalar_t invscale = (scale >= 1.0) ? 1.0 / scale : 1.0;
  accscalar_t center = scale * (i + 0.5);
  xmin = max(
      static_cast<int64_t>(center - support + 0.5), static_cast<int64_t>(0));
  xmax = min(static_cast<int64_t>(center + support + 0.5), input_size) - xmin;

  accscalar_t total_w = 0.0;
  int64_t j = 0;
  for (j = 0; j < xmax; j++) {
    accscalar_t w = filter_fn((j + xmin - center + 0.5) * invscale);
    wt_ptr[j] = static_cast<scalar_t>(w);
    total_w += w;
  }
  for (j = 0; j < xmax; j++) {
    if (total_w != 0.0) {
      wt_ptr[j] /= total_w;
    }
  }
  for (; j < interp_size; j++) {
    wt_ptr[j] = static_cast<scalar_t>(0.0);
  }
}

template <typename scalar_t, typename accscalar_t>
__device__ __forceinline__ static accscalar_t interpolate_aa_single_dim(
    scalar_t* src,
    scalar_t* weights,
    int64_t size) {
  scalar_t t = static_cast<accscalar_t>(*src);
  scalar_t wts = static_cast<accscalar_t>(weights[0]);
  accscalar_t output = t * wts;

  int64_t j = 1;
  for (; j < size; j++) {
    wts = static_cast<accscalar_t>(weights[j]);
    t = static_cast<accscalar_t>(*(src + j));
    output += t * wts;
  }
  return output;
}

template <typename scalar_t, typename accscalar_t, int interp_size>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_gen2d_out_frame(
    const int n,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    const PackedTensorAccessor64<scalar_t, 4> idata,
    PackedTensorAccessor64<scalar_t, 4> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int height1 = idata.size(2);
  const int width1 = idata.size(3);
  const int height2 = odata.size(2);
  const int width2 = odata.size(3);

  if (index < n) {
    const int w2 = index % width2; // 0:width2-1
    const int h2 = index / width2; // 0:height2-1
    // special case: just copy
    if (height1 == height2 && width1 == width2) {
      const int h1 = h2;
      const int w1 = w2;
      for (int n = 0; n < batchsize; n++) {
        for (int c = 0; c < channels; ++c) {
          const scalar_t val = idata[n][c][h1][w1];
          odata[n][c][h2][w2] = val;
        }
      }
      return;
    }

    const accscalar_t support_h = static_cast<accscalar_t>(
        (rheight >= 1.0) ? (interp_size * 0.5) * rheight : interp_size * 0.5);
    const accscalar_t support_w = static_cast<accscalar_t>(
        (rwidth >= 1.0) ? (interp_size * 0.5) * rwidth : interp_size * 0.5);

    const int interp_height = (int)ceilf(support_h) * 2 + 1;
    const int interp_width = (int)ceilf(support_w) * 2 + 1;

    // Setup local buffers
    // TODO: maybe we can specify dynamic shared memory size before calling the
    // cuda code, however we should then ensure that device has enough shared
    // memory
    scalar_t wx[256];
    scalar_t wy[256];
    scalar_t buffer1[256];
    scalar_t buffer2[256];

    // Compute weights
    int64_t xmin, xsize, ymin, ysize;
    typedef scalar_t (*filter_fn_t)(scalar_t);
    if (interp_size == 2) {
      _compute_weights<scalar_t, accscalar_t, filter_fn_t>(
          w2,
          width1,
          rwidth,
          support_w,
          wx,
          interp_width,
          bilinear_filter,
          xmin,
          xsize);
      _compute_weights<scalar_t, accscalar_t, filter_fn_t>(
          h2,
          height1,
          rheight,
          support_h,
          wy,
          interp_height,
          bilinear_filter,
          ymin,
          ysize);
    } else if (interp_size == 4) {
      _compute_weights<scalar_t, accscalar_t, filter_fn_t>(
          w2,
          width1,
          rwidth,
          support_w,
          wx,
          interp_width,
          bicubic_filter,
          xmin,
          xsize);
      _compute_weights<scalar_t, accscalar_t, filter_fn_t>(
          h2,
          height1,
          rheight,
          support_h,
          wy,
          interp_height,
          bicubic_filter,
          ymin,
          ysize);
    }

    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        // interpolate on x-axis for ymin to ymin + ysize
        for (int64_t y = 0; y < ysize; y++) {
          // copy data into the local buffer and use
          // interpolate_aa_single_dim method
          for (int x = 0; x < xsize; x++) {
            buffer1[x] = idata[n][c][ymin + y][xmin + x];
          }

          buffer2[y] = static_cast<scalar_t>(
              interpolate_aa_single_dim<scalar_t, accscalar_t>(
                  buffer1, wx, xsize));
        }
        odata[n][c][h2][w2] = static_cast<scalar_t>(
            interpolate_aa_single_dim<scalar_t, accscalar_t>(
                buffer2, wy, ysize));
      }
    }
  }
}

template <int interp_size>
static void upsample_gen2d_out_cuda_template(
    const Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  TensorArg input_arg{input, "input", 1}, output_arg{output, "output", 2};
  checkAllSameGPU("upsample_gen2d_out_cuda", {input_arg, output_arg});

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input.size(0);
  int channels = input.size(1);
  int input_height = input.size(2);
  int input_width = input.size(3);

  const int num_kernels = output_height * output_width;
  const int num_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "upsample_bilinear2d_out_frame", [&] {
        using accscalar_t = at::acc_type<scalar_t, true>;

        auto idata = input.packed_accessor64<scalar_t, 4>();
        auto odata = output.packed_accessor64<scalar_t, 4>();

        const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
            input_height, output_height, align_corners, scales_h);
        const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
            input_width, output_width, align_corners, scales_w);

        // We are using static buffer memory of 256 * sizeof(float) per thread
        // to store weights. Size of weights array is
        // interp_size = scale * 2 + 1 for bilinear mode
        TORCH_CHECK(
            rheight < (255 / interp_size),
            "Max supported scale factor is 127 (bilinear), 63 (bicubic)");
        TORCH_CHECK(
            rwidth < (255 / interp_size),
            "Max supported scale factor is 127 (bilinear), 63 (bicubic)");

        upsample_gen2d_out_frame<scalar_t, accscalar_t, interp_size>
            <<<cuda::ATenCeilDiv(num_kernels, num_threads),
               num_threads,
               0,
               stream>>>(
                num_kernels, rheight, rwidth, align_corners, idata, odata);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
}

} // namespace internal_upsample
} // namespace native
} // namespace at

namespace vision {
namespace ops {

namespace {

// Copied from "UpSample.h" as we can not use UpSample.h with UpSample.cuh
static std::array<int64_t, 4> upsample_2d_common_check(
    at::IntArrayRef input_size,
    at::IntArrayRef output_size) {
  TORCH_CHECK(
      output_size.size() == 2,
      "It is expected output_size equals to 2, but got size ",
      output_size.size());

  TORCH_CHECK(
      input_size.size() == 4,
      "It is expected input_size equals to 4, but got size ",
      input_size.size());

  int64_t output_height = output_size[0];
  int64_t output_width = output_size[1];

  int64_t nbatch = input_size[0];
  int64_t channels = input_size[1];
  int64_t input_height = input_size[2];
  int64_t input_width = input_size[3];

  TORCH_CHECK(
      input_height > 0 && input_width > 0 && output_height > 0 &&
          output_width > 0,
      "Input and output sizes should be greater than 0,"
      " but got input (H: ",
      input_height,
      ", W: ",
      input_width,
      ") output (H: ",
      output_height,
      ", W: ",
      output_width,
      ")");

  return {nbatch, channels, output_height, output_width};
}

template <int interp_size>
at::Tensor interpolate_gen2d_aa_forward_kernel(
    const at::Tensor& input,
    at::IntArrayRef output_size,
    bool align_corners) {
  c10::optional<c10::ArrayRef<double>> scale_factors = {};

  // Copied from UpSampleBilinear2d.cpp
  auto output = at::empty({0}, input.options());
  auto osize = at::native::upsample::compute_output_size(
      input.sizes(), output_size, scale_factors);
  auto scale_h = at::native::upsample_cuda::get_scale_value(scale_factors, 0);
  auto scale_w = at::native::upsample_cuda::get_scale_value(scale_factors, 1);

  auto full_output_size = upsample_2d_common_check(input.sizes(), osize);

  // Allow for empty batch size but not other dimensions
  TORCH_CHECK(
      input.numel() != 0 ||
          c10::multiply_integers(
              input.sizes().begin() + 1, input.sizes().end()),
      "Non-empty 4D data tensor expected but got a tensor with sizes ",
      input.sizes());

  output.resize_(full_output_size, input.suggest_memory_format());

  at::native::internal_upsample::upsample_gen2d_out_cuda_template<interp_size>(
      output,
      input,
      {full_output_size[2], full_output_size[3]},
      align_corners,
      scale_h,
      scale_w);
  return output;
}

at::Tensor interpolate_linear_aa_forward_kernel(
    const at::Tensor& input,
    at::IntArrayRef output_size,
    bool align_corners) {
  return interpolate_gen2d_aa_forward_kernel<2>(
      input, output_size, align_corners);
}

at::Tensor interpolate_bicubic_aa_forward_kernel(
    const at::Tensor& input,
    at::IntArrayRef output_size,
    bool align_corners) {
  return interpolate_gen2d_aa_forward_kernel<4>(
      input, output_size, align_corners);
}

} // namespace

TORCH_LIBRARY_IMPL(torchvision, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("torchvision::_interpolate_linear_aa"),
      TORCH_FN(interpolate_linear_aa_forward_kernel));
  m.impl(
      TORCH_SELECTIVE_NAME("torchvision::_interpolate_bicubic_aa"),
      TORCH_FN(interpolate_bicubic_aa_forward_kernel));
}

} // namespace ops
} // namespace vision
